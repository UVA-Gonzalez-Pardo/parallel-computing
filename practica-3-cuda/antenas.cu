#include "hip/hip_runtime.h"
/**
 * Computación Paralela (curso 1516)
 *
 * Colocación de antenas
 * Versión Cuda
 *
 * @author Daniel González Alonso
 * @author Santos Pardo Ramos
 */

// Includes generales
#include <stdio.h>
#include <stdlib.h>
#include <limits.h>

// CUDA
#include <hip/hip_runtime.h>

// Include para las utilidades de computación paralela
#include <cutil.h>


/**
 * Estructura antena
 */
typedef struct {
    int y;
    int x;
    int valor;
} Antena;

/**
 * Macro para acceder a las posiciones del mapa
 */
#define m(y,x) mapa[ ((y) * cols) + x ]


#define NUM_THREADS_PER_BLOCK       512
#define NUM_BLOCKS                  4
#define NUM_THREADS                 (NUM_THREADS_PER_BLOCK * NUM_BLOCKS)

//Declaración del vector resultados de la reducción
static int* d_Result;




/**
 * Función de ayuda para imprimir el mapa
 */
__global__ void print_mapa(int* mapa, int rows, int cols, Antena* a)
{
    if(rows > 50 || cols > 30){
        printf("Mapa muy grande para imprimir\n");
        return;
    }

    #define ANSI_COLOR_RED     "\x1b[31m"
    #define ANSI_COLOR_GREEN   "\x1b[32m"
    #define ANSI_COLOR_RESET   "\x1b[0m"

    printf("Mapa [%d,%d]\n", rows, cols);
    for(int i = 0; i < rows; i++) {
        for(int j = 0; j < cols; j++)
        {
            int val = m(i, j);

            if(val == 0){
                if(a != NULL && a->x == j && a->y == i){
                    printf( ANSI_COLOR_RED "   A"  ANSI_COLOR_RESET);
                } else { 
                    printf( ANSI_COLOR_GREEN "   A"  ANSI_COLOR_RESET);
                }
            } else {
                printf("%4d", val);
            }
        }// j
        printf("\n");
    }// i
    printf("\n");
}


/**
 * Distancia de una antena a un punto (y,x)
 * @note Es el cuadrado de la distancia para tener más carga
 */
__device__ void manhattan(Antena* a, int y, int x, int* resultado)
{
    int dist = abs(a->x - x) + abs(a->y - y);
    (*resultado) = dist * dist;
}


/**
 * Actualizar el mapa con la nueva antena
 */
__global__ void actualizar(int* mapa, int rows, int cols, Antena* antena)
{
    // El bloque es un cuadrado 2x2, si estamos en el hilo (0,0) actualizamos
    // la antena.
    if (threadIdx.x == 0 && threadIdx.y == 0)
        m(antena->y, antena->x) = 0;

    int r       = 1,
        flag    = -1,
        distancia[1];

    while (flag != 0)
    {
        flag = 0;       // Reiniciamos la bandera

        manhattan(antena, antena->y - r, antena->x, &distancia[0]);

        if (threadIdx.x == 0 && threadIdx.y == 0)
        {
            Antena izquierda = (Antena){ antena->y, (antena->x - r), -1 };

            // Si izquierda.x esta fuera del mapa, iniciamos delta en la
            // primera interseccion del rombo con el mapa por la izquierda.
            int delta = (izquierda.x < 0)? -(izquierda.x) : 0;

            // Ascendemos por el lateral superior izquierdo del rombo
            for (; (delta < r) && (izquierda.y - delta >= 0); delta++)
            {
                if (m(izquierda.y - delta, izquierda.x + delta) > distancia[0])
                {
                    //printf("IDBX:%d, IDBY:%d dist=%d, aby=%d, abx=%d delta:%d, r:%d\n", threadIdx.x, threadIdx.y, distancia[0], izquierda.y - delta, izquierda.x + delta, delta, r);
                    m(izquierda.y - delta, izquierda.x + delta) = distancia[0];
                    flag = 1;
                }
            }
        }

        if (threadIdx.x == 1 && threadIdx.y == 0)
        {
            Antena arriba = (Antena){ (antena->y - r), antena->x, -1 };

            // Si arriba.y esta fuera del mapa, iniciamos delta en la primera
            // interseccion del rombo con el mapa por arriba.
            int delta = (arriba.y < 0)? -(arriba.y) : 0;
            
            // Descendemos por el lateral superior derecho del rombo
            for (; (delta < r) && (arriba.x + delta < cols); delta++)
            {
                if (m(arriba.y + delta, arriba.x + delta) > distancia[0])
                {
                    //printf("IDBX:%d, IDBY:%d dist=%d, aby=%d, abx=%d delta:%d, r:%d\n", threadIdx.x, threadIdx.y, distancia[0], arriba.y + delta, arriba.x + delta, delta, r);
                    m(arriba.y + delta, arriba.x + delta) = distancia[0];
                    flag = 1;
                }
            }
        }

        if (threadIdx.x == 1 && threadIdx.y == 1)
        {
            Antena derecha = (Antena){ antena->y, (antena->x + r), -1 };

            // Si derecha.x esta fuera del mapa, iniciamos delta en la primera
            // interseccion del rombo con el mapa por la derecha.
            int delta = (derecha.x >= cols)? derecha.x + 1 - cols : 0;
            
            // Descendemos por el lateral inferior derecho del rombo
            for (; (delta < r) && (derecha.y + delta < rows); delta++)
            {
                if (m(derecha.y + delta, derecha.x - delta) > distancia[0])
                {
                    //printf("IDBX:%d, IDBY:%d dist=%d, aby=%d, abx=%d delta:%d, r:%d\n", threadIdx.x, threadIdx.y, distancia[0], derecha.y + delta, derecha.x - delta, delta, r);
                    m(derecha.y + delta, derecha.x - delta) = distancia[0];
                    flag = 1;
                }
            }
        }

        if (threadIdx.x == 0 && threadIdx.y == 1)
        {
            Antena abajo = (Antena){ (antena->y + r), antena->x, -1 };

            // Si abajo.y esta fuera del mapa, iniciamos delta en la primera
            // interseccion del rombo con el mapa por abajo.
            int delta = (abajo.y >= rows)? abajo.y + 1 - rows : 0;

            // Ascendemos por el lateral inferior izquierdo del rombo
            for (; (delta < r) && (abajo.x - delta >= 0); delta++)
            {
                if (m(abajo.y - delta, abajo.x - delta) > distancia[0])
                {
                    //printf("IDBX:%d, IDBY:%d dist=%d, aby=%d, abx=%d delta:%d, r:%d\n", threadIdx.x, threadIdx.y, distancia[0], abajo.y - delta, abajo.x - delta, delta, r);
                    m(abajo.y - delta, abajo.x - delta) = distancia[0];
                    flag = 1;
                }
            }
        }

        r++;
    }

    __syncthreads();
}


/**
 * Actualizar el mapa con la primera antena
 */
__global__ void actualizar_primera_antena(int* mapa, int rows, int cols,
    Antena* antena)
{
    // El bloque es un cuadrado 2x2, si estamos en el hilo (0,0) actualizamos
    // la antena.
    if (threadIdx.x == 0 && threadIdx.y == 0)
        m(antena->y, antena->x) = 0;

    int r       = 1,
        flag    = -1,
        distancia[1];

    while (flag != 0)
    {
        flag = 0;       // Reiniciamos la bandera

        manhattan(antena, antena->y - r, antena->x, &distancia[0]);

        if (threadIdx.x == 0 && threadIdx.y == 0)
        {
            Antena izquierda = (Antena){ antena->y, (antena->x - r), -1 };

            // Si izquierda.x esta fuera del mapa, iniciamos delta en la
            // primera interseccion del rombo con el mapa por la izquierda.
            int delta = (izquierda.x < 0)? -(izquierda.x) : 0;

            // Ascendemos por el lateral superior izquierdo del rombo
            for (; (delta < r) && (izquierda.y - delta >= 0); delta++)
            {
                m(izquierda.y - delta, izquierda.x + delta) = distancia[0];
                flag = 1;
            }
        }

        if (threadIdx.x == 1 && threadIdx.y == 0)
        {
            Antena arriba = (Antena){ (antena->y - r), antena->x, -1 };

            // Si arriba.y esta fuera del mapa, iniciamos delta en la primera
            // interseccion del rombo con el mapa por arriba.
            int delta = (arriba.y < 0)? -(arriba.y) : 0;
            
            // Descendemos por el lateral superior derecho del rombo
            for (; (delta < r) && (arriba.x + delta < cols); delta++)
            {
                m(arriba.y + delta, arriba.x + delta) = distancia[0];
                flag = 1;
            }
        }

        if (threadIdx.x == 1 && threadIdx.y == 1)
        {
            Antena derecha = (Antena){ antena->y, (antena->x + r), -1 };

            // Si derecha.x esta fuera del mapa, iniciamos delta en la primera
            // interseccion del rombo con el mapa por la derecha.
            int delta = (derecha.x >= cols)? derecha.x + 1 - cols : 0;
            
            // Descendemos por el lateral inferior derecho del rombo
            for (; (delta < r) && (derecha.y + delta < rows); delta++)
            {
                m(derecha.y + delta, derecha.x - delta) = distancia[0];
                flag = 1;
            }
        }

        if (threadIdx.x == 0 && threadIdx.y == 1)
        {
            Antena abajo = (Antena){ (antena->y + r), antena->x, -1 };

            // Si abajo.y esta fuera del mapa, iniciamos delta en la primera
            // interseccion del rombo con el mapa por abajo.
            int delta = (abajo.y >= rows)? abajo.y + 1 - rows : 0;

            // Ascendemos por el lateral inferior izquierdo del rombo
            for (; (delta < r) && (abajo.x - delta >= 0); delta++)
            {
                m(abajo.y - delta, abajo.x - delta) = distancia[0];
                flag = 1;
            }
        }

        r++;
    }

    __syncthreads();
}


/**
 * Kernel que realiza la reducción de un array de entrada y lo deja en un array de salida
 */
__global__ void reduce_kernel(const int* g_idata, int numValues, int* g_odata){
    extern __shared__ int sdata[];

    // cada hilo carga un elemento desde memoria global hacia memoria shared
    unsigned int tid = threadIdx.x;
    unsigned int igl = blockIdx.x;
    sdata[tid] = g_idata[blockDim.x * igl + tid];
    __syncthreads();
    
    if ((blockDim.x * igl + tid) <= numValues)
    {
        // Hacemos la reducción en memoria shared
        for(unsigned int s = 1; s < blockDim.x; s *= 2) {
            // Comprobamos si el hilo actual es activo para esta iteración
            if (tid % (2*s) == 0){
                // Hacemos la reducción sumando los dos elementos que le tocan a este hilo
                if (sdata[tid] < sdata[tid + s])
                    sdata[tid] = sdata[tid + s];
            }
            __syncthreads();
        }
    }

    // El hilo 0 de cada bloque escribe el resultado final de la reducción
    // en la memoria global del dispositivo pasada por parámetro (g_odata[])
    if (tid == 0)
        g_odata[blockIdx.x] = sdata[0];

}


/**
 * Función que se encarga de lanzar los kernels para realizar la reducción
 * del maximo
 */
extern "C" int* reduce(const int* values, unsigned int numValues){

    int numThreadsPerBlock = NUM_THREADS_PER_BLOCK;
    int numBlocks = NUM_BLOCKS;

    //La primera pasada reduce el array de entrada: VALUES
    //a un array de igual tamaño que el número total de bloques del grid: D_RESULT
    int sharedMemorySize = numThreadsPerBlock * sizeof(int);

    reduce_kernel<<<numBlocks, numThreadsPerBlock, sharedMemorySize>>>(values, numValues, d_Result);

    //La segunda pasada lanza sólo un único bloque para realizar la reducción final
    numThreadsPerBlock = numBlocks;
    numBlocks = 1;
    sharedMemorySize = numThreadsPerBlock * sizeof(int);
    reduce_kernel<<<numBlocks, numThreadsPerBlock, sharedMemorySize>>>(d_Result, numValues, d_Result);

    return d_Result;
}


/**
 * Calcular la posición de la nueva antena
 */
__global__ void nueva_antena(int* mapa, int rows, int cols, int min, Antena* maximo)
{
    for(int i=0; i<rows; i++){
        for(int j=0; j<cols; j++){

            if(m(i,j)==min)
            {
                (*maximo) = (Antena){ i, j, min };
            }

        } // j
    } // i

}



/**
 * Función principal
 */
int main(int nargs, char** vargs)
{
    //
    // 1. LEER DATOS DE ENTRADA
    //
    
    // Comprobar número de argumentos
    if (nargs < 7)
    {
        fprintf(stderr,
                "Uso: %s rows cols distMax nAntenas x0 y0 [x1 y1, ...]\n",
                vargs[0]);
        return -1;
    }

    // Leer los argumentos de entrada
    int rows = atoi(vargs[1]);
    int cols = atoi(vargs[2]);
    int distMax = atoi(vargs[3]);
    int nAntenas = atoi(vargs[4]);

    if (nAntenas<1 || nargs != (nAntenas*2+5))
    {
        fprintf(stderr, "Error en la lista de antenas\n");
        return -1;
    }

    // Mensaje
    printf("Calculando el número de antenas necesarias para cubrir un"
        " mapa de (%d x %d)\ncon una distancia máxima no superior a %d "
        "y con %d antenas iniciales\n\n", rows, cols, distMax, nAntenas);

    // Reservar memoria para las antenas
    Antena* antenas = (Antena*)malloc(sizeof(Antena) * nAntenas);
    if (!antenas)
    {
        fprintf( stderr,
                "Error al reservar memoria para las antenas inicales\n" );
        return -1;
    }
    
    // Leer antenas
    for (int i = 0; i < nAntenas; i++)
    {
        antenas[i].x = atoi(vargs[5+i*2]);
        antenas[i].y = atoi(vargs[6+i*2]);

        if( antenas[i].y < 0 || antenas[i].y >= rows
            || antenas[i].x < 0 || antenas[i].x >= cols )
        {
            fprintf(stderr, "Antena #%d está fuera del mapa\n", i);
            return -1;
        }
    }

    //
    // 2. INICIACIÓN
    //

    // Medir el tiempo
    double tiempo = cp_Wtime();


    unsigned int numValues = rows * cols;

    // Antena sobre la que trabajaremos en el device
    Antena* antenaDevice;
    hipMalloc((void**)&antenaDevice, sizeof(Antena));

    // Array sobre el que haremos la reduccion en reduce_kernel
    hipMalloc((void**)&d_Result, NUM_BLOCKS * sizeof(int));

    // Crear el mapa
    int* mapa;
    hipMalloc((void**)&mapa, rows * cols * sizeof(int)) ;

    // Declaración del shape de los bloques y del grid
    dim3 gridShape1(1);
    dim3 bloqShape1(2, 2);


    // Colocar las antenas iniciales
    hipMemcpy(antenaDevice, &antenas[0], sizeof(Antena), hipMemcpyHostToDevice);
    actualizar_primera_antena<<<gridShape1, bloqShape1>>>(mapa, rows, cols, antenaDevice);
    for (int i = 1; i < nAntenas; i++)
    {
        hipMemcpy(antenaDevice, &antenas[i], sizeof(Antena), hipMemcpyHostToDevice) ;
        actualizar<<<gridShape1, bloqShape1>>>(mapa, rows, cols, antenaDevice);
    }

    // Debug
#ifdef DEBUG
    print_mapa(mapa, rows, cols, NULL);
#endif


    //
    // 3. CALCULO DE LAS NUEVAS ANTENAS
    //
    
    int nuevas = 0;     // Contador de antenas
    int* valor = (int*)malloc( sizeof(int)) ;

    while (1)
    {
        // Calcular la posicion de la antena a distancia maxima.
        reduce(mapa, numValues);

        hipMemcpy(valor, d_Result, sizeof(int), hipMemcpyDeviceToHost) ;

        // Salimos si ya hemos cumplido el maximo
        if ((*valor) <= distMax) break;

        // Incrementamos el contador
        nuevas++;

        nueva_antena<<<1,1>>>(mapa, rows, cols, (*valor), antenaDevice);

        // Actualización del mapa con la nueva antena y nuevas distancias.
        actualizar<<<gridShape1, bloqShape1>>>(mapa, rows, cols, antenaDevice);
    }

    // Debug
#ifdef DEBUG
    print_mapa(mapa, rows, cols, NULL);
#endif


    //
    // 4. MOSTRAR RESULTADOS
    //

    // tiempo
    tiempo = cp_Wtime() - tiempo;   

    // Salida
    printf("Result: %d\n", nuevas);
    printf("Time: %f\n", tiempo);
    //print_mapa<<<1, 1>>>(mapa, rows, cols, NULL);

    // Liberamos memoria del device
    hipFree(mapa);
    hipFree(antenaDevice);
    hipFree(d_Result);
    
    // Liberamos memoria del HOST
    free(antenas);
    free(valor);

    // Liberamos los hilos del DEVICE
    hipDeviceReset();

    return 0;
}
